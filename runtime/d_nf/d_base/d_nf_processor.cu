#include "hip/hip_runtime.h"
#include "d_network_function_base.cuh"
#include "d_network_function_derived.cuh"
#include "d_nf_item.h"
#include "../d_firewall/d_firewall.cuh"
#include "../d_flowmonitor/d_flow_monitor.cuh"
#include "../d_httpparser/d_http_parser.cuh"
#include "../d_pktcounter/d_pkt_counter.cuh"
#include "Pkt.h"
#include "d_nf_processor.cuh"

__device__ void Init_nfs(struct d_flow_actor_nfs* nfs){


	nfs->nf[1]=new d_network_function_derived<d_pkt_counter, d_pkt_counter_fs>(1);
	nfs->nf[2]=new d_network_function_derived<d_flow_monitor, d_flow_monitor_fs>(2);
	nfs->nf[3]=new d_network_function_derived<d_firewall, d_firewall_fs>(3);
	nfs->nf[4]=new d_network_function_derived<d_http_parser, d_http_parser_fs>(4);

}



__device__ uint8_t compute_network_function(uint64_t s, int pos){
  return static_cast<uint8_t>((s>>(8*pos))&0x00000000000000FF);
}

__device__ int compute_service_chain_length(uint64_t s){
  int length = 0;
  bool encounter_zero = false;
  for(int i=0; i<8; i++){
    uint8_t nf =
        static_cast<uint8_t>((s>>(8*i))&0x00000000000000FF);
    if(nf>0){
      length+=1;
      if(encounter_zero){
        return -1;
      }
    }
    else{
      encounter_zero = true;
    }
  }
  return length;
}


__global__ void
Runtask(Pkt* pkts, Fs* fs, uint64_t service_chain,int packet_num)
{


	struct d_flow_actor_nfs  nfs;
	Init_nfs(&nfs);
	int chain_len=compute_service_chain_length(service_chain);
	int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < packet_num)
    {
    	int j=i;
    	while(pkts[j].empty!=true){
    		for(int k=0; k<chain_len; k++){
    			int nf_id=compute_network_function(service_chain,k);
    			nfs.nf[nf_id]->nf_logic(pkts[j].pkt,fs[j%packet_num].fs[nf_id]);
    		}
    		j+=packet_num;

    	}
    }
}



void gpu_nf_process(Pkt* pkts,Fs* fs,uint64_t service_chain,int packet_num){

    int threadsPerBlock = 256;
    int blocksPerGrid =(packet_num + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    Runtask<<<blocksPerGrid, threadsPerBlock>>>(pkts, fs, service_chain, packet_num);
    hipDeviceSynchronize();


}


